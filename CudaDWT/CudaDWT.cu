#include "hip/hip_runtime.h"
#include <vector>
using namespace std;
#include <stdio.h>
#include <assert.h>
#include "libclock.h"
#include "liblog.h"
#include "cuda_macro.h"
#include "CudaDWT.h"

inline const char* SZGetCudppError(const CUDPPResult result)
{
	switch(result)
	{
	case CUDPP_ERROR_INVALID_HANDLE:
		return	"Specified handle (for example, to a plan) is invalid.";
	case CUDPP_ERROR_ILLEGAL_CONFIGURATION:
		return	"Specified configuration is illegal. For example, an invalid or illogical combination of options. ";
	case CUDPP_ERROR_INVALID_PLAN:
		return	"The plan is not configured properly. For example, passing a plan for scan to cudppSegmentedScan. ";
	case CUDPP_ERROR_INSUFFICIENT_RESOURCES:
		return	"The function could not complete due to insufficient resources (typically CUDA device resources such as shared memory) for the specified problem size. ";
	case CUDPP_ERROR_UNKNOWN:
		return	"Unknown or untraceable error.";
	}
	return "No Error";
}

#define ASSERT_CUDPP(call)	\
	{	\
		CUDPPResult result = call;	\
		if( CUDPP_SUCCESS != result )	\
		{	\
			CUT_CHECK_ERROR(# call);	\
			LOG_ERROR(cerr<<SZGetCudppError(result));	\
		}	\
	}	\
	

#define BLOCK_SIZE_X	16
#define BLOCK_SIZE_Y	8
#define BLOCK_SIZE		(BLOCK_SIZE_X * BLOCK_SIZE_Y)

#include "ProjToWavelet_kernel.h"
#include "MarkSegments_kernel.h"

namespace CudaDWT
{
	CCudaDWT::
		~CCudaDWT
	(
	)
	{
		if( bIsInitialized )
		{
			// free the memory space
			FREE_MEMORY(pu4BinSub_device);
			FREE_MEMORY(pfValues_device);
			FREE_MEMORY(puKeys_device);
			FREE_MEMORY(puiSegFlags_device);
			FREE_MEMORY(pfCoefs_device);
			FREE_MEMORY(pfCoefSums_device);
			FREE_MEMORY(pfCompactedCoefs_device);
			FREE_MEMORY(puCompactedKeys_device);

			// ADD-BY-LEETEN 01/13/2013-BEGIN
			FREE_MEMORY(puOnes_device);
			FREE_MEMORY(puSegCounts_device);
			FREE_MEMORY(puCompactedSegCounts_device);
			FREE_MEMORY(puNrOfCompactedSegCounts_device);
			// ADD-BY-LEETEN 01/13/2013-END

			// ADD-BY-LEETEN 01/11/2013-BEGIN
			#if		WITH_CUDA_MALLOC_HOST	
			FREE_MEMORY_ON_HOST(puNrOfCompactedKeys_host);
			FREE_MEMORY_ON_HOST(puNrOfCompactedCoefs_host);
			FREE_MEMORY_ON_HOST(pfCoefs_host);
			FREE_MEMORY_ON_HOST(puKeys_host);
			#endif	// #if		WITH_CUDA_MALLOC_HOST	
			// ADD-BY-LEETEN 01/11/2013-END

			FREE_MEMORY(puNrOfCompactedCoefs_device);
			FREE_MEMORY(puNrOfCompactedKeys_device);

			// free cudpp resources
			if( planSort )
				ASSERT_CUDPP(cudppDestroyPlan(planSort));  
			if( planSegScanCoefs )
				ASSERT_CUDPP(cudppDestroyPlan(planSegScanCoefs));
			if( planCompactCoefs )
				ASSERT_CUDPP(cudppDestroyPlan(planCompactCoefs));
			if( planCompactKeys )
				ASSERT_CUDPP(cudppDestroyPlan(planCompactKeys));
			// ADD-BY-LEETEN 01/13/2013-BEGIN
			if( planSegScanCounts )
				ASSERT_CUDPP(cudppDestroyPlan(planSegScanCounts));

			if( planCompactSegCounts )
				ASSERT_CUDPP(cudppDestroyPlan(planCompactSegCounts));
			// ADD-BY-LEETEN 01/13/2013-END
			if( theCudpp )
				ASSERT_CUDPP(cudppDestroy(theCudpp));
		}
	}

	void
	CCudaDWT::
	_Init
	(
		size_t* puMaxNrOfElementsOnTheDevice,
		void* _Reserved
	)
	{
		ASSERT_CUDPP(cudppCreate(&theCudpp));
		
		if( *puMaxNrOfElementsOnTheDevice > CudaDWT::DEFAULT_MAX_NR_OF_ELEMENTS_ON_THE_DEVICE )
		{
			LOG_ERROR(cerr<<"uMaxNrOfElementsOnTheDevice is clampped to CudaDWT::DEFAULT_MAX_NR_OF_ELEMENTS_ON_THE_DEVICE");
			*puMaxNrOfElementsOnTheDevice = CudaDWT::DEFAULT_MAX_NR_OF_ELEMENTS_ON_THE_DEVICE;
		}

		size_t uMaxNrOfElementsOnTheDevice = *puMaxNrOfElementsOnTheDevice;

		configSort.op = CUDPP_ADD;
		configSort.datatype = CUDPP_UINT;
		configSort.algorithm = CUDPP_SORT_RADIX;
		configSort.options = CUDPP_OPTION_KEY_VALUE_PAIRS;
		ASSERT_CUDPP(cudppPlan(theCudpp, &planSort, configSort, uMaxNrOfElementsOnTheDevice, 1, 0));  

		configSegScanCoefs.op = CUDPP_ADD;
		configSegScanCoefs.datatype = CUDPP_FLOAT;
		configSegScanCoefs.algorithm = CUDPP_SEGMENTED_SCAN;
		configSegScanCoefs.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;
		ASSERT_CUDPP(cudppPlan(theCudpp, &planSegScanCoefs, configSegScanCoefs, uMaxNrOfElementsOnTheDevice, 1, 0));

		configCompactCoefs.datatype = CUDPP_FLOAT;
		configCompactCoefs.algorithm = CUDPP_COMPACT;
		configCompactCoefs.options = CUDPP_OPTION_FORWARD;
		ASSERT_CUDPP(cudppPlan(theCudpp, &planCompactCoefs, configCompactCoefs, uMaxNrOfElementsOnTheDevice, 1, 0));

		configCompactKeys.datatype = CUDPP_UINT;
		configCompactKeys.algorithm = CUDPP_COMPACT;
		configCompactKeys.options = CUDPP_OPTION_FORWARD;
		ASSERT_CUDPP(cudppPlan(theCudpp, &planCompactKeys, configCompactKeys, uMaxNrOfElementsOnTheDevice, 1, 0));

		// allocate the memory space
		CUDA_SAFE_CALL(hipMalloc((void**)&pu4BinSub_device,		sizeof(pu4BinSub_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&pfValues_device,			sizeof(pfValues_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&puKeys_device,			sizeof(puKeys_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&puiSegFlags_device,		sizeof(puiSegFlags_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&pfCoefs_device,			sizeof(pfCoefs_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&pfCoefSums_device,		sizeof(pfCoefSums_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&pfCompactedCoefs_device,	sizeof(pfCompactedCoefs_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&puCompactedKeys_device,	sizeof(puCompactedKeys_device[0]) * uMaxNrOfElementsOnTheDevice));

		// ADD-BY-LEETEN 01/13/2013-BEGIN
		configSegScanCounts.datatype = CUDPP_UINT;
		configSegScanCounts.algorithm = CUDPP_SEGMENTED_SCAN;
		configSegScanCounts.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;
		ASSERT_CUDPP(cudppPlan(theCudpp, &planSegScanCounts, configSegScanCounts, uMaxNrOfElementsOnTheDevice, 1, 0));

		configCompactSegCounts.datatype = CUDPP_UINT;
		configCompactSegCounts.algorithm = CUDPP_COMPACT;
		configCompactSegCounts.options = CUDPP_OPTION_FORWARD;
		ASSERT_CUDPP(cudppPlan(theCudpp, &planCompactSegCounts, configCompactSegCounts, uMaxNrOfElementsOnTheDevice, 1, 0));

		CUDA_SAFE_CALL(hipMalloc((void**)&puOnes_device,			sizeof(puOnes_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&puSegCounts_device,		sizeof(puSegCounts_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&puCompactedSegCounts_device,			sizeof(puCompactedSegCounts_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&puNrOfCompactedSegCounts_device,		sizeof(puNrOfCompactedSegCounts_device[0])));

		vector<unsigned int> vuOnes;
		vuOnes.assign(uMaxNrOfElementsOnTheDevice, 1);
		CUDA_SAFE_CALL(hipMemcpy(puOnes_device, vuOnes.data(), sizeof(puOnes_device[0]) * vuOnes.size(), hipMemcpyHostToDevice));
		// ADD-BY-LEETEN 01/13/2013-END

		// ADD-BY-LEETEN 01/11/2013-BEGIN
		#if		WITH_CUDA_MALLOC_HOST	
		CUDA_SAFE_CALL(hipHostMalloc((void**)&puNrOfCompactedKeys_host,			sizeof(puNrOfCompactedKeys_host[0])));
		CUDA_SAFE_CALL(hipHostMalloc((void**)&puNrOfCompactedCoefs_host,			sizeof(puNrOfCompactedCoefs_host[0])));
		CUDA_SAFE_CALL(hipHostMalloc((void**)&puKeys_host,			sizeof(puKeys_host[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipHostMalloc((void**)&pfCoefs_host,		sizeof(pfCoefs_host[0]) * uMaxNrOfElementsOnTheDevice));
		#endif	// #if	WITH_CUDA_MALLOC_HOST	
		// ADD-BY-LEETEN 01/11/2013-END

		CUDA_SAFE_CALL(hipMalloc((void**)&puNrOfCompactedCoefs_device,	sizeof(puNrOfCompactedCoefs_device[0]) * uMaxNrOfElementsOnTheDevice));
		CUDA_SAFE_CALL(hipMalloc((void**)&puNrOfCompactedKeys_device,	sizeof(puNrOfCompactedKeys_device[0]) * uMaxNrOfElementsOnTheDevice));

		bIsInitialized = true;
	}

	void
	CCudaDWT::
	_InitEncoder
	(
		// ADD-BY-LEETEN 01/18/2012-BEGIN
		size_t uNrOfDims,
		unsigned int puCoefLengths[],
		// ADD-BY-LEETEN 01/18/2012-END
		size_t				uNrOfElements,
		const uint4			pu4BinSubs[],
		const float			pfValues[],
		bool bWithCpuBucketSort,	// ADD-BY-LEETEN 01/13/2013
		void* _Reserved
	)
	{
		this->bWithCpuBucketSort = bWithCpuBucketSort;	// ADD-BY-LEETEN 01/13/2012

		// upload the tuples in the pool to the device side
		CUDA_SAFE_CALL(
			hipMemcpy(
				&pu4BinSub_device[0], 
				&pu4BinSubs[0],		
				sizeof(pu4BinSub_device[0]) * uNrOfElements, 
				hipMemcpyHostToDevice));

		CUDA_SAFE_CALL(
			hipMemcpy(
				&pfValues_device[0],	
				&pfValues[0],
				sizeof(pfValues_device[0]) * uNrOfElements, 
				hipMemcpyHostToDevice));

		// ADD-BY-LEETEN 01/18/2012-BEGIN
		CUDA_SAFE_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(
				puCoefLengths_const), 
				puCoefLengths,
				uNrOfDims * sizeof(puCoefLengths_const[0]), 
				0,
				hipMemcpyHostToDevice));
		// ADD-BY-LEETEN 01/18/2012-END

		// ADD-BY-LEETEN 01/11/2013-BEGIN
		v3Blk = dim3(BLOCK_SIZE);
		size_t uNrOfBlocks = (size_t)ceilf((float)uNrOfElements / (float)v3Blk.x);
		size_t uGridSizeX = (size_t)ceil(sqrtf((float)uNrOfBlocks));
		size_t uGridSizeY = (size_t)ceil((float)uNrOfBlocks/(float)uGridSizeX);
		// MOD-BY-LEETEN 01/12/2013-FROM:		v3Grid = dim3(uGridSizeX, uGridSizeY);
		v3Grid = dim3( (unsigned int)uGridSizeX, (unsigned int)uGridSizeY );
		// MOD-BY-LEETEN 01/12/2013-END
		// ADD-BY-LEETEN 01/11/2013-END
	}

	void
	CCudaDWT::
	_Encode
	(
		size_t				uNrOfElements,
		size_t				uNrOfDims,
		const unsigned int	puLevels[],
		const unsigned int	puWaveletLengths[],

		size_t				*puNrOfElements,
		#if		!WITH_CUDA_MALLOC_HOST	// ADD-BY-LEETEN 01/11/2013
		unsigned int		puKeys_host[],
		float				pfCoefs_host[],
		// ADD-BY-LEETEN 01/11/2013-BEGIN
		#else	// #if		!WITH_CUDA_MALLOC_HOST
		unsigned int		puKeys[],
		float				pfCoefs[],
		#endif	// #if		!WITH_CUDA_MALLOC_HOST
		// ADD-BY-LEETEN 01/11/2013-END

		unsigned int		puSegCounts_host[],	// ADD-BY-LEETEN 01/13/2013

		int iTimingPrintingLevel,	// ADD-BY-LEETEN 01/11/2013
		void* _Reserved
	)
	{
		bool bIsPrintingTiming = (iTimingPrintingLevel > 0)?true:false;	// ADD-BY-LEETEN 01/11/2013
		LIBCLOCK_INIT(bIsPrintingTiming, __FUNCTION__);

		// copy the lengths of the local coefficient array, wavelet lengths, and levels
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		#if	0	// MOD-BY-LEETEN 01/11/2013-FROM:
		CUDA_SAFE_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(
				"puLevels_const"), 
				&puLevels[0], 
				sizeof(puLevels_const[0]) * uNrOfDims,
				0,
				hipMemcpyHostToDevice) );

		CUDA_SAFE_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(
				"puWaveletLengths_const"), 
				&puWaveletLengths[0], 
				sizeof(puWaveletLengths_const[0]) * uNrOfDims, 
				0,
				hipMemcpyHostToDevice) );
		#else	// MOD-BY-LEETEN 01/11/2013-TO:
		CUDA_SAFE_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(
				puLevels_const), 
				&puLevels[0], 
				sizeof(puLevels_const[0]) * uNrOfDims,
				0,
				hipMemcpyHostToDevice) );

		CUDA_SAFE_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(
				puWaveletLengths_const), 
				&puWaveletLengths[0], 
				sizeof(puWaveletLengths_const[0]) * uNrOfDims, 
				0,
				hipMemcpyHostToDevice) );
		#endif	// MOD-BY-LEETEN 01/11/2013-END
		LIBCLOCK_END(bIsPrintingTiming);

		LIBCLOCK_BEGIN(bIsPrintingTiming);
		// 
		#if	0	// DEL-BY-LEETEN 01/11/2013-BEGIN
		dim3 v3Blk = dim3(BLOCK_SIZE);
		dim3 v3Grid = dim3((size_t)ceilf((float)uNrOfElements / (float)v3Blk.x));
		#endif	// DEL-BY-LEETEN 01/11/2013-END

		_ProjToWavelet_kernel<<<v3Grid, v3Blk, 0>>>(
			&pu4BinSub_device[0],	// the tuples of <bin, data_subscripts> of all elements
			&pfValues_device[0],	// the counts of all elements
			(unsigned int)uNrOfDims, 
			(unsigned int)uNrOfElements,
			&puKeys_device[0],		// output: the keys of all elements. The keys are composed of bin and local_subscripts
			&pfCoefs_device[0]		// output: the wavelet projection of the current wavelet 
			);
		CUT_CHECK_ERROR("_ProjToWavelet_kernel() failed");
		LIBCLOCK_END(bIsPrintingTiming);

		// ADD-BY-LEETEN 01/13/2013-BEGIN
		if( !bWithCpuBucketSort )	
		{
		// ADD-BY-LEETEN 01/13/2013-END
		// sort the wavelet projection according to the key composed by the bin and local subscripts
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		ASSERT_CUDPP(cudppSort(
			planSort,				
			&puKeys_device[0],		
			&pfCoefs_device[0],
			uNrOfElements));
		LIBCLOCK_END(bIsPrintingTiming);
		}	// ADD-BY-LEETEN 01/13/2013

		// mark the segments. the beginning of a segment is marked as 1, and all other elements are marked as 0
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		_MarkSegments_kernel<<<v3Grid, v3Blk, 0>>>(
			&puKeys_device[0],
			(unsigned int)uNrOfElements,
			&puiSegFlags_device[0]);
		CUT_CHECK_ERROR("_MarkSegments_kernel() failed");
		LIBCLOCK_END(bIsPrintingTiming);

		// ADD-BY-LEETEN 01/13/2013-BEGIN
		// compute the count per segment
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		ASSERT_CUDPP(cudppSegmentedScan(
			planSegScanCounts,
			&puSegCounts_device[0],
			&puOnes_device[0],
			&puiSegFlags_device[0],
			uNrOfElements));
		LIBCLOCK_END(bIsPrintingTiming);

		// compact the result
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		ASSERT_CUDPP(cudppCompact(
			planCompactSegCounts,
			&puCompactedSegCounts_device[0],
			puNrOfCompactedSegCounts_device,
			&puSegCounts_device[0],
			&puiSegFlags_device[0],
			uNrOfElements));
		LIBCLOCK_END(bIsPrintingTiming);

		LIBCLOCK_BEGIN(bIsPrintingTiming);
		size_t uNrOfCompactedSegCounts_host;
		CUDA_SAFE_CALL(
			hipMemcpy(
				&uNrOfCompactedSegCounts_host, 
				puNrOfCompactedSegCounts_device, 
				sizeof(uNrOfCompactedSegCounts_host), 
				hipMemcpyDeviceToHost));
		LIBCLOCK_END(bIsPrintingTiming);

		LIBCLOCK_BEGIN(bIsPrintingTiming);
		CUDA_SAFE_CALL(
			hipMemcpy(
				&puSegCounts_host[0],
				&puCompactedSegCounts_device[0], 
				uNrOfCompactedSegCounts_host * sizeof(puSegCounts_host[0]),
				hipMemcpyDeviceToHost) );
		LIBCLOCK_END(bIsPrintingTiming);
		// ADD-BY-LEETEN 01/13/2013-END

		// compute the sum of the segments
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		ASSERT_CUDPP(cudppSegmentedScan(
			planSegScanCoefs,
			&pfCoefSums_device[0],
			&pfCoefs_device[0],
			&puiSegFlags_device[0],
			uNrOfElements));
		LIBCLOCK_END(bIsPrintingTiming);

		// compact the result
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		size_t uNrOfCompactedCoefs_host;
		ASSERT_CUDPP(cudppCompact(
			planCompactCoefs,
			&pfCompactedCoefs_device[0],
			puNrOfCompactedCoefs_device,
			&pfCoefSums_device[0],
			&puiSegFlags_device[0],
			uNrOfElements));
		LIBCLOCK_END(bIsPrintingTiming);

		LIBCLOCK_BEGIN(bIsPrintingTiming);
		#if		!WITH_CUDA_MALLOC_HOST	// ADD-BY-LEETEN 01/11/2013
		CUDA_SAFE_CALL(
			hipMemcpy(
				&uNrOfCompactedCoefs_host, 
				puNrOfCompactedCoefs_device, 
				sizeof(uNrOfCompactedCoefs_host), 
				hipMemcpyDeviceToHost));
		// ADD-BY-LEETEN 01/11/2013-BEGIN
		#else	// #if		!WITH_CUDA_MALLOC_HOST	
		CUDA_SAFE_CALL(
			hipMemcpy(
				puNrOfCompactedCoefs_host, 
				puNrOfCompactedCoefs_device, 
				sizeof(uNrOfCompactedCoefs_host), 
				hipMemcpyDeviceToHost));
		uNrOfCompactedCoefs_host = *puNrOfCompactedCoefs_host;
		#endif	// #if		!WITH_CUDA_MALLOC_HOST	
		// ADD-BY-LEETEN 01/11/2013-END
		LIBCLOCK_END(bIsPrintingTiming);

		LIBCLOCK_BEGIN(bIsPrintingTiming);
		CUDA_SAFE_CALL(
			hipMemcpy(
				&pfCoefs_host[0],
				&pfCompactedCoefs_device[0], 
				uNrOfCompactedCoefs_host * sizeof(pfCoefs_host[0]),
				hipMemcpyDeviceToHost) );
		// ADD-BY-LEETEN 01/11/2013-BEGIN
		#if		WITH_CUDA_MALLOC_HOST	
		memcpy(&pfCoefs[0], &pfCoefs_host[0], uNrOfCompactedCoefs_host * sizeof(pfCoefs[0]));
		#endif	// #if		WITH_CUDA_MALLOC_HOST	
		// ADD-BY-LEETEN 01/11/2013-END
		LIBCLOCK_END(bIsPrintingTiming);

		// compact the keys
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		size_t uNrOfCompactedKeys_host;
		ASSERT_CUDPP(cudppCompact(
			planCompactKeys,
			&puCompactedKeys_device[0],
			puNrOfCompactedKeys_device,
			&puKeys_device[0],
			&puiSegFlags_device[0],
			uNrOfElements));
		LIBCLOCK_END(bIsPrintingTiming);

		LIBCLOCK_BEGIN(bIsPrintingTiming);
		#if		!WITH_CUDA_MALLOC_HOST	// ADD-BY-LEETEN 01/11/2013
		CUDA_SAFE_CALL(
			hipMemcpy(
				&uNrOfCompactedKeys_host, 
				puNrOfCompactedKeys_device, 
				sizeof(uNrOfCompactedKeys_host), 
				hipMemcpyDeviceToHost));
		// ADD-BY-LEETEN 01/11/2013-BEGIN
		#else	// #if		!WITH_CUDA_MALLOC_HOST	
		CUDA_SAFE_CALL(
			hipMemcpy(
				puNrOfCompactedKeys_host, 
				puNrOfCompactedKeys_device, 
				sizeof(uNrOfCompactedKeys_host), 
				hipMemcpyDeviceToHost));
		uNrOfCompactedKeys_host = *puNrOfCompactedKeys_host;
		#endif	// #if		!WITH_CUDA_MALLOC_HOST	
		// ADD-BY-LEETEN 01/11/2013-END
		LIBCLOCK_END(bIsPrintingTiming);

		// download the keys and the coefficinets back 
		LIBCLOCK_BEGIN(bIsPrintingTiming);
		CUDA_SAFE_CALL(
			hipMemcpy(
				&puKeys_host[0],
				&puCompactedKeys_device[0], 
				uNrOfCompactedKeys_host * sizeof(puKeys_host[0]),
				hipMemcpyDeviceToHost) );
		// ADD-BY-LEETEN 01/11/2013-BEGIN
		#if		WITH_CUDA_MALLOC_HOST	
		memcpy(&puKeys[0], &puKeys_host[0], uNrOfCompactedKeys_host * sizeof(puKeys[0]));
		#endif	// #if		WITH_CUDA_MALLOC_HOST	
		// ADD-BY-LEETEN 01/11/2013-END
		LIBCLOCK_END(bIsPrintingTiming);

		ASSERT_OR_LOG(uNrOfCompactedKeys_host == uNrOfCompactedCoefs_host, cerr<<"Unmatched #keys and #coefs.");
		*puNrOfElements = uNrOfCompactedKeys_host;
		LIBCLOCK_PRINT(bIsPrintingTiming);
	}
};

